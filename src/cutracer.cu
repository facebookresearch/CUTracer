#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) Meta Platforms, Inc. and affiliates.
 * SPDX-FileCopyrightText: Copyright (c) 2019 NVIDIA CORPORATION & AFFILIATES.
 * SPDX-License-Identifier: MIT AND BSD-3-Clause
 *
 * This source code contains modifications by Meta Platforms, Inc. licensed under MIT,
 * based on original NVIDIA NVBit sample code licensed under BSD-3-Clause.
 * See LICENSE file in the root directory for Meta's license terms.
 * See LICENSE-BSD file in the root directory for NVIDIA's license terms.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

#include <map>
#include <string>
#include <unordered_map>
#include <unordered_set>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* contains definition of the reg_info_t and mem_access_t structure */
#include "common.h"

/* analysis functionality */
#include "analysis.h"

/* env config */
#include "env_config.h"

/* logging functionality */
#include "log.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)

/* lock */
pthread_mutex_t mutex;
pthread_mutex_t cuda_event_mutex;

/* map to store context state */
std::unordered_map<hipCtx_t, CTXstate *> ctx_state_map;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_callback_flag = false;

std::map<int, std::string> id_to_sass_map;
/* grid launch id, incremented at every launch */
uint64_t global_grid_launch_id = 0;

// map to store the iteration count for each kernel
static std::map<hipFunction_t, uint32_t> kernel_iter_map;


/* ===== Main Functionality ===== */
// Based on NVIDIA NVBit record_reg_vals and mem_trace examples with Meta modifications for unified register support
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  CTXstate *ctx_state = ctx_state_map[ctx];

  /* Get related functions of the kernel (device function that can be
   * called by the kernel) */
  std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);

  /* add kernel itself to the related function vector */
  related_functions.push_back(func);

  /* iterate on function */
  for (auto f : related_functions) {
    const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
    if (verbose) {
      loprintf("Inspecting function %s at address 0x%lx\n", nvbit_get_func_name(ctx, f), nvbit_get_func_addr(ctx, f));
    }

    uint32_t cnt = 0;
    /* iterate on all the static instructions in the function */
    for (auto instr : instrs) {
      if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
          instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
          instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
        cnt++;
        continue;
      }
      if (verbose) {
        instr->printDecoded();
      }

      std::vector<int> reg_num_list;
      std::vector<int> ureg_num_list;
      int mref_idx = 0;
      int opcode_id = instr->getIdx();
      id_to_sass_map[opcode_id] = std::string(instr->getSass());
      /* iterate on the operands */
      for (int i = 0; i < instr->getNumOperands(); i++) {
        /* get the operand "i" */
        const InstrType::operand_t *op = instr->getOperand(i);
        if (op->type == InstrType::OperandType::REG) {
          for (int reg_idx = 0; reg_idx < instr->getSize() / 4; reg_idx++) {
            reg_num_list.push_back(op->u.reg.num + reg_idx);
          }
        } else if (op->type == InstrType::OperandType::UREG) {
          for (int reg_idx = 0; reg_idx < instr->getSize() / 4; reg_idx++) {
            ureg_num_list.push_back(op->u.reg.num + reg_idx);
          }
        } else if (op->type == InstrType::OperandType::MREF) {
          // TODO: double check this with NVIDIA people
          if (op->u.mref.has_desc) {
            ureg_num_list.push_back(op->u.mref.desc_ureg_num);
            ureg_num_list.push_back(op->u.mref.desc_ureg_num + 1);
          }
          /* insert call to the instrumentation function with its
           * arguments */
          nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
          /* predicate value */
          nvbit_add_call_arg_guard_pred_val(instr);
          /* opcode id */
          nvbit_add_call_arg_const_val32(instr, opcode_id);
          /* memory reference 64 bit address */
          nvbit_add_call_arg_mref_addr64(instr, mref_idx);
          /* add "space" for kernel function pointer that will be set
           * at launch time (64 bit value at offset 0 of the dynamic
           * arguments)*/
          nvbit_add_call_arg_launch_val64(instr, 0);
          /* add instruction PC */
          nvbit_add_call_arg_const_val64(instr, instr->getOffset());
          /* add pointer to channel_dev*/
          nvbit_add_call_arg_const_val64(instr, (uint64_t)ctx_state->channel_dev);
          mref_idx++;
        }
      }

      /* insert call to the instrumentation function with its arguments */
      nvbit_insert_call(instr, "record_reg_val", IPOINT_BEFORE);
      /* guard predicate value */
      nvbit_add_call_arg_guard_pred_val(instr);
      /* opcode id */
      nvbit_add_call_arg_const_val32(instr, opcode_id);
      /* add pointer to channel_dev*/
      nvbit_add_call_arg_const_val64(instr, (uint64_t)ctx_state->channel_dev);
      /* add instruction PC */
      nvbit_add_call_arg_const_val64(instr, instr->getOffset());
      /* how many register values are passed next */
      nvbit_add_call_arg_const_val32(instr, reg_num_list.size());
      nvbit_add_call_arg_const_val32(instr, ureg_num_list.size());
      for (int num : reg_num_list) {
        /* last parameter tells it is a variadic parameter passed to
         * the instrument function record_reg_val() */
        nvbit_add_call_arg_reg_val(instr, num, true);
      }
      for (int num : ureg_num_list) {
        nvbit_add_call_arg_ureg_val(instr, num, true);
      }
    }
  }
}

// Reference code from NVIDIA nvbit mem_trace tool
/* flush channel */
__global__ void flush_channel(ChannelDev *ch_dev) { ch_dev->flush(); }

// Reference code from NVIDIA nvbit mem_trace tool
void init_context_state(hipCtx_t ctx) {
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  CTXstate *ctx_state = ctx_state_map[ctx];
  ctx_state->recv_thread_done = RecvThreadState::WORKING;
  hipMallocManaged(&ctx_state->channel_dev, sizeof(ChannelDev));
  ctx_state->channel_host.init((int)ctx_state_map.size() - 1, CHANNEL_SIZE, ctx_state->channel_dev, recv_thread_fun,
                               ctx);
  nvbit_set_tool_pthread(ctx_state->channel_host.get_thread());
}
// Reference code from NVIDIA nvbit mem_trace tool
static void enter_kernel_launch(hipCtx_t ctx, hipFunction_t func, uint64_t &grid_launch_id, nvbit_api_cuda_t cbid,
                                void *params, bool stream_capture = false, bool build_graph = false) {
  // no need to sync during stream capture or manual graph build, since no
  // kernel is actually launched.
  if (!stream_capture && !build_graph) {
    /* Make sure GPU is idle */
    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);
  }

  instrument_function_if_needed(ctx, func);

  int nregs = 0;
  CUDA_SAFECALL(hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func));

  int shmem_static_nbytes = 0;
  CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, func));

  /* get function name and pc */
  const char *func_name = nvbit_get_func_name(ctx, func);
  uint64_t pc = nvbit_get_func_addr(ctx, func);

  // during stream capture or manual graph build, no kernel is launched, so
  // do not set launch argument, do not print kernel info, do not increase
  // grid_launch_id. All these should be done at graph node launch time.
  if (!stream_capture && !build_graph) {
    /* set grid launch id at launch time */
    nvbit_set_at_launch(ctx, func, (uint64_t)grid_launch_id);

    if (cbid == API_CUDA_cuLaunchKernelEx_ptsz || cbid == API_CUDA_cuLaunchKernelEx) {
      cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
      loprintf(
          "MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - "
          "Kernel name %s - grid launch id %ld - grid size %d,%d,%d "
          "- block size %d,%d,%d - nregs %d - shmem %d - cuda stream "
          "id %ld\n",
          (uint64_t)ctx, pc, func_name, grid_launch_id, p->config->gridDimX, p->config->gridDimY, p->config->gridDimZ,
          p->config->blockDimX, p->config->blockDimY, p->config->blockDimZ, nregs,
          shmem_static_nbytes + p->config->sharedMemBytes, (uint64_t)p->config->hStream);
    } else {
      cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
      loprintf(
          "MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - "
          "Kernel name %s - grid launch id %ld - grid size %d,%d,%d "
          "- block size %d,%d,%d - nregs %d - shmem %d - cuda stream "
          "id %ld\n",
          (uint64_t)ctx, pc, func_name, grid_launch_id, p->gridDimX, p->gridDimY, p->gridDimZ, p->blockDimX,
          p->blockDimY, p->blockDimZ, nregs, shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
    }

    // increment grid launch id for next launch
    // grid id can be changed here, since nvbit_set_at_launch() has copied
    // its value above.
    grid_launch_id++;
  }

  /* enable instrumented code to run */
  nvbit_enable_instrumented(ctx, func, true);
}

// the function is only called for non cuda graph launch cases.
static void leave_kernel_launch(CTXstate *ctx_state, uint64_t &grid_launch_id) {
  // make sure user kernel finishes to avoid deadlock
  hipDeviceSynchronize();
  /* push a flush channel kernel */
  flush_channel<<<1, 1>>>(ctx_state->channel_dev);

  /* Make sure GPU is idle */
  hipDeviceSynchronize();
  assert(hipGetLastError() == hipSuccess);
}
// Reference code from NVIDIA nvbit mem_trace tool
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid, const char *name, void *params,
                         hipError_t *pStatus) {
  pthread_mutex_lock(&cuda_event_mutex);

  /* we prevent re-entry on this callback when issuing CUDA functions inside
   * this function */
  if (skip_callback_flag) {
    pthread_mutex_unlock(&cuda_event_mutex);
    return;
  }
  skip_callback_flag = true;

  CTXstate *ctx_state = ctx_state_map[ctx];

  switch (cbid) {
    // Identify all the possible CUDA launch events without stream
    // parameters, they will not get involved with cuda graph
    case API_CUDA_cuLaunch:
    case API_CUDA_cuLaunchGrid: {
      cuLaunch_params *p = (cuLaunch_params *)params;
      hipFunction_t func = p->f;
      if (!is_exit) {
        enter_kernel_launch(ctx, func, global_grid_launch_id, cbid, params);
      } else {
        leave_kernel_launch(ctx_state, global_grid_launch_id);
      }
    } break;
    // To support kernel launched by cuda graph (in addition to existing kernel
    // launche method), we need to do:
    //
    // 1. instrument kernels at hipGraphAddKernelNode event. This is for cases
    // that kernels are manually added to a cuda graph.
    // 2. distinguish captured kernels when kernels are recorded to a graph
    // using stream capture. hipStreamIsCapturing() tells us whether a stream
    // is capturiong.
    // 3. per-kernel instruction counters, since cuda graph can launch multiple
    // kernels at the same time.
    //
    // Three cases:
    //
    // 1. original kernel launch:
    //     1a. for any kernel launch without using a stream, we instrument it
    //     before it is launched, call hipDeviceSynchronize after it is
    //     launched and read the instruction counter of the kernel.
    //     1b. for any kernel launch using a stream, but the stream is not
    //     capturing, we do the same thing as 1a.
    //
    //  2. cuda graph using stream capturing: if a kernel is launched in a
    //  stream and the stream is capturing. We instrument the kernel before it
    //  is launched and do nothing after it is launched, because the kernel is
    //  not running until hipGraphLaunch. Instead, we issue a
    //  hipStreamSynchronize after hipGraphLaunch is done and reset the
    //  instruction counters, since a cloned graph might be launched afterwards.
    //
    //  3. cuda graph manual: we instrument the kernel added by
    //  hipGraphAddKernelNode and do the same thing for hipGraphLaunch as 2.
    //
    // The above method should handle most of cuda graph launch cases.
    // kernel launches with stream parameter, they can be used for cuda graph
    case API_CUDA_cuLaunchKernel_ptsz:
    case API_CUDA_cuLaunchKernel:
    case API_CUDA_cuLaunchCooperativeKernel:
    case API_CUDA_cuLaunchCooperativeKernel_ptsz:
    case API_CUDA_cuLaunchKernelEx:
    case API_CUDA_cuLaunchKernelEx_ptsz:
    case API_CUDA_cuLaunchGridAsync: {
      hipFunction_t func;
      hipStream_t hStream;

      if (cbid == API_CUDA_cuLaunchKernelEx_ptsz || cbid == API_CUDA_cuLaunchKernelEx) {
        cuLaunchKernelEx_params *p = (cuLaunchKernelEx_params *)params;
        func = p->f;
        hStream = p->config->hStream;
      } else if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel ||
                 cbid == API_CUDA_cuLaunchCooperativeKernel_ptsz || cbid == API_CUDA_cuLaunchCooperativeKernel) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;
        func = p->f;
        hStream = p->hStream;
      } else {
        cuLaunchGridAsync_params *p = (cuLaunchGridAsync_params *)params;
        func = p->f;
        hStream = p->hStream;
      }

      hipStreamCaptureStatus streamStatus;
      /* check if the stream is capturing, if yes, do not sync */
      CUDA_SAFECALL(hipStreamIsCapturing(hStream, &streamStatus));
      if (!is_exit) {
        bool stream_capture = (streamStatus == hipStreamCaptureStatusActive);
        enter_kernel_launch(ctx, func, global_grid_launch_id, cbid, params, stream_capture);
        log_open_kernel_file(ctx, func, kernel_iter_map[func]++);
      } else {
        if (streamStatus != hipStreamCaptureStatusActive) {
          if (verbose >= 1) {
            loprintf("kernel %s not captured by cuda graph\n", nvbit_get_func_name(ctx, func));
          }
          leave_kernel_launch(ctx_state, global_grid_launch_id);
        } else {
          if (verbose >= 1) {
            loprintf("kernel %s captured by cuda graph\n", nvbit_get_func_name(ctx, func));
          }
        }
      }
    } break;
    case API_CUDA_cuGraphAddKernelNode: {
      cuGraphAddKernelNode_params *p = (cuGraphAddKernelNode_params *)params;
      hipFunction_t func = p->nodeParams->func;

      if (!is_exit) {
        // cuGraphAddKernelNode_params->nodeParams is the same as
        // cuLaunchKernel_params up to sharedMemBytes
        enter_kernel_launch(ctx, func, global_grid_launch_id, cbid, (void *)p->nodeParams, false, true);
      }
    } break;
    case API_CUDA_cuGraphLaunch: {
      // if we are exiting a cuda graph launch:
      // Wait until the graph is completed using
      // hipStreamSynchronize()
      if (is_exit) {
        cuGraphLaunch_params *p = (cuGraphLaunch_params *)params;

        CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
        assert(hipGetLastError() == hipSuccess);
        /* push a flush channel kernel */
        flush_channel<<<1, 1, 0, p->hStream>>>(ctx_state->channel_dev);
        CUDA_SAFECALL(hipStreamSynchronize(p->hStream));
        assert(hipGetLastError() == hipSuccess);
      }

    } break;
    default:
      break;
  };

  skip_callback_flag = false;
  pthread_mutex_unlock(&cuda_event_mutex);
}

// Reference NVIDIA record_reg_vals example
void nvbit_tool_init(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  init_context_state(ctx);
  pthread_mutex_unlock(&mutex);
}

// Reference code from NVIDIA nvbit mem_trace tool
void nvbit_at_ctx_init(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  if (verbose) {
    printf("MEMTRACE: STARTING CONTEXT %p\n", ctx);
  }
  assert(ctx_state_map.find(ctx) == ctx_state_map.end());
  CTXstate *ctx_state = new CTXstate;
  ctx_state_map[ctx] = ctx_state;
  pthread_mutex_unlock(&mutex);
}

// Reference code from NVIDIA nvbit mem_trace tool
void nvbit_at_ctx_term(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  skip_callback_flag = true;
  if (verbose) {
    loprintf("MEMTRACE: TERMINATING CONTEXT %p\n", ctx);
  }
  /* get context state from map */
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  CTXstate *ctx_state = ctx_state_map[ctx];

  /* Notify receiver thread and wait for receiver thread to
   * notify back */
  ctx_state->recv_thread_done = RecvThreadState::STOP;
  while (ctx_state->recv_thread_done != RecvThreadState::FINISHED);

  ctx_state->channel_host.destroy(false);
  hipFree(ctx_state->channel_dev);
  skip_callback_flag = false;
  delete ctx_state;
  pthread_mutex_unlock(&mutex);
  // Cleanup log handle system
  cleanup_log_handle();
}

// Reference code from NVIDIA nvbit mem_trace tool
void nvbit_at_graph_node_launch(hipCtx_t ctx, hipFunction_t func, hipStream_t stream, uint64_t launch_handle) {
  func_config_t config = {0};
  const char *func_name = nvbit_get_func_name(ctx, func);
  uint64_t pc = nvbit_get_func_addr(ctx, func);

  pthread_mutex_lock(&mutex);
  nvbit_set_at_launch(ctx, func, (uint64_t)global_grid_launch_id, stream, launch_handle);
  nvbit_get_func_config(ctx, func, &config);

  loprintf(
      "MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - "
      "Kernel name %s - grid launch id %ld - grid size %d,%d,%d "
      "- block size %d,%d,%d - nregs %d - shmem %d - cuda stream "
      "id %ld\n",
      (uint64_t)ctx, pc, func_name, global_grid_launch_id, config.gridDimX, config.gridDimY, config.gridDimZ,
      config.blockDimX, config.blockDimY, config.blockDimZ, config.num_registers,
      config.shmem_static_nbytes + config.shmem_dynamic_nbytes, (uint64_t)stream);
  // grid id can be changed here, since nvbit_set_at_launch() has copied its
  // value above.
  global_grid_launch_id++;
  pthread_mutex_unlock(&mutex);
}

// Reference code from NVIDIA nvbit mem_trace tool with Meta modifications for env config
void nvbit_at_init() {
  init_log_handle();
  // Initialize configuration from environment variables
  init_config_from_env();
  /* set mutex as recursive */
  pthread_mutexattr_t attr;
  pthread_mutexattr_init(&attr);
  pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
  pthread_mutex_init(&mutex, &attr);

  pthread_mutex_init(&cuda_event_mutex, &attr);
}
